#include "hip/hip_runtime.h"
extern "C" {

    /**
     * cube_interp: subeic interpolation assuming regular spacing
     */
    float cube_interp(
        const float x0,     // starting position of the regularly spaced coordinate vector
        const float dx,     // spacing of the coordinate vector
        const int nx,       // number of points in the coordinate vector
        const float* y,     // values of of the function sampled on x
        float xout,         // coordintate to interpolate at
        float fill          // value to fill if out of bounds
    ) 
    {
        float xn = x0 + dx * (nx-1);
        
        // boundary condition (bc) - exactly last sampled point
        if (xout == xn) return y[nx-1];

        // bc - out of bounds, use fill value
        else if ((xout < x0) || (xout > xn)) return fill;
                                
        int ixo = (int) ((xout- x0)/dx);
        float mm2, mm1, mp0, mp1, mp2, w0, w1, sp0, sp1, a, b, c, d, delta;

        // bc - first point
        if (ixo == 0) {
            mp0 = y[ixo+1] - y[ixo+0];
            mp1 = y[ixo+2] - y[ixo+1];

            sp0 = mp0;
            sp1 = (mp0 + mp1)/2;

        } 
        // bc - second point
        else if (ixo == 1) {
            mm1 = y[ixo+0] - y[ixo-1];
            mp0 = y[ixo+1] - y[ixo-0];
            mp1 = y[ixo+2] - y[ixo+1];
            mp2 = y[ixo+3] - y[ixo+2];

            sp0 = (mp0 + mm1)/2;

            if ((mm1 == mp0) && (mp0 == mp1) && (mp1 == mp2)) sp1 = 0;
            else {
                w0  = abs(mp2 - mp1) + abs(mp2 + mp1)/2;
                w1  = abs(mp0 - mm1) + abs(mp0 + mm1)/2;
                sp1 = (w0 * mp0 + w1 * mp1) / (w0 + w1);
            }
            

        } 
        // bc - third to last point
        else if (ixo == nx-3) {
            mm2 = y[ixo+-1] - y[ixo-2];
            mm1 = y[ixo+0] - y[ixo-1];
            mp0 = y[ixo+1] - y[ixo-0];
            mp1 = y[ixo+2] - y[ixo+1];

            if ((mm2 == mm1) && (mm1 == mp0) && (mp0 == mp1)) sp0 = 0;
            else {
                w0  = abs(mp1 - mp0) + abs(mp1 + mp0)/2;
                w1  = abs(mm1 - mm2) + abs(mm1 + mm2)/2;
                sp0 = (w0 * mm1 + w1*mp0) / (w0 + w1);
            }

            sp1 = (mp0 + mp1)/2;
        }
        // bc - second to last point
        else if (ixo == nx-2) {
            mm1 = y[ixo+0] - y[ixo-1];
            mp0 = y[ixo+1] - y[ixo+0];

            sp0 = (mm1+mp0)/2;

            sp1 = mp0;
        }
        // all other points
        else {
            mm2 = y[ixo-1] - y[ixo-2];
            mm1 = y[ixo+0] - y[ixo-1];
            mp0 = y[ixo+1] - y[ixo+0];
            mp1 = y[ixo+2] - y[ixo+1];
            mp2 = y[ixo+3] - y[ixo+2];

            if ((mm2 == mm1) && (mm1 == mp0) && (mp0 == mp1)) sp0 = 0;
            else {
                w0  = abs(mp1 - mp0) + abs(mp1 + mp0)/2;
                w1  = abs(mm1 - mm2) + abs(mm1 + mm2)/2;
                sp0 = (w0 * mm1 + w1*mp0) / (w0 + w1);
            }

            if ((mm1 == mp0) && (mp0 == mp1) && (mp1 == mp2)) sp1 = 0;
            else {
                w0  = abs(mp2 - mp1) + abs(mp2 + mp1)/2;
                w1  = abs(mp0 - mm1) + abs(mp0 + mm1)/2;
                sp1 = (w0 * mp0 + w1*mp1) / (w0 + w1);
            }
        }

        a = y[ixo];
        b = sp0;
        c = (3*mp0 - 2*sp0 - sp1)/dx;
        d = (sp0 + sp1 - 2*mp0)/(dx*dx);

        delta = xout - (x0 + dx * ixo);

        return a + b * delta + c * delta * delta + d * delta * delta * delta;
    }

    /**
     * xInfo: struct defining the bounds and spacing of a regularly spaced array
     */
    struct xInfo {
        float x0;   // the starting point of the vector
        float dx;   // the spacing between points
        int nx;     // the number of points in the vector
    };

    /**
     * RFInfo: struct defining the meadata of RF data
     */
    struct RFInfo {
        int ntx;            // the number of transmit events
        int nrx;            // the number of recieve events
        int ndim;           // the number of dimensions to beamform over
        struct xInfo tInfo; // the sampling information about the time vector
    };

    /**
     * calc_tautx_apodtx: calcualte the tx delay tabs and apodizations given transmit data structures
     * 
     * tau and apod are pointers to be filled with the correct values
     */
    void calc_tautx_apodtx(
        const int    ndim,  // 2 or 3 dimensions
        const float*  foc,  // focal spot
        const float* nvec,  // normal vector of wave propagation
        const float    c0,  // assumed speed of sound in media
        const float    t0,  // the time at which the wave reaches foc
        const float   ala,  // the acceptance angle relative to nvec - zero for plane wave
        const float   dof,  // the dof around foc over which to "flatten" the delay tabs
        const float* pvec,  // the point at which we are calculating delay tabs and apodization
        float* tau, float* apod // output numbers
    )
    {
        
        // calculate the magnitude of dx and its projection onto nvec
        float dxi, dxmag, dxproj;
        dxmag  = 0.0;
        dxproj = 0.0;
        for (int idim = 0; idim < ndim; ++idim)
        {
            dxi = pvec[idim] - foc[idim];
            dxmag  += dxi * dxi;
            dxproj += dxi * nvec[idim];
        }
        dxmag = sqrt(dxmag);

        // if synthetic focal point (diverging or converging waves)
        if (0.0 != ala)
        {
            if ((dof > 1E-9) && (abs(dxproj) <= dof/2)) {
                *tau = 2.0*(dxproj/dof)*(dxmag/c0) + t0;
                if (sqrt(abs(dxmag*dxmag - dxproj*dxproj)) <= dof * sin(ala) / 2.0) *apod = 1.0;
                else *apod = 0.0;
            }
            else {
                if (abs(dxproj) > 1E-9) *tau = (dxproj/abs(dxproj)) * (dxmag/c0) + t0;
                else *tau = t0;
                if ((abs(dxmag) > 1E-9) && (acos(abs(dxproj/dxmag)) > ala)) *apod = 0.0;
                else *apod = 1.0;
            }
        }

        // plane wave case
        else {
            *tau = dxproj/c0;
            *apod = 1.0;
        }
    }

    /**
     * calc_taurx_apodrx: calcualte the rx delay tabs and apodizations given recieve data structures
     * 
     * tau and apod are pointers to be filled with the correct values
     */
    void calc_taurx_apodrx(
        const int    ndim,  // 2 or 3 dimensions
        const float* orig,  // origin of receive element
        const float* nvec,  // normal vector of wave propagation
        const float    c0,  // assumed speed of sound in media
        const float   ala,  // the acceptance angle relative to nvec - zero for plane wave
        const float* pvec,  // the point at which we are calculating delay tabs and apodization
        float* tau, float* apod // output numbers
    )
    {
        // calculate the magnitude of dx and its projection onto nvec
        float dxi, dxmag, dxproj;
        dxmag  = 0.0;
        dxproj = 0.0;
        for (int idim = 0; idim < ndim; ++idim)
        {
            dxi = pvec[idim] - orig[idim];
            dxmag  += dxi * dxi;
            dxproj += dxi * nvec[idim];
        }
        dxmag = sqrt(dxmag);

        // calculate receive delay tabs and apodization
        *tau = dxmag/c0;
        if ((dxmag != 0.0) && (acos(abs(dxproj/dxmag)) > ala)) *apod = 0.0;
        else *apod = 1.0;
    }


    /**
     * das_bmode_cubic: beamform a DAS bmode 
     * 
     * RF channel data parameters:
     *   rfinfo: information about the rfdata
     *   rfdata: grid of rf data in the shape ntx by nrx by nt (stored in tInfo)
     * 
     * Transmit parameters:
     *   ovectx: origin of each transmision
     *   nvectx: normal vector of each of the transmit events
     *   t0tx: the timepoint at which the wave is at ovectx in each transmit event
     *   alatx: acceptance angle in radians relative to each nvectx
     * 
     * Receive parameters:
     *   ovecrx: origin of the receive points
     *   nvecrx: normal vector of each receive element
     *   alarx:  angular acceptance for each element relative to nvecrx
     * 
     * Field parameters:
     *   c0: the homogeneos speed of sound in the medium
     *   np: the number of recon points
     *   pvec: the location of each recon point
     *   pout: a vector length p for the output bmode
     */
    __global__ 
    void das_bmode_cubic(
        const struct RFInfo rfinfo, const float* rfdata, 
        const float* ovectx, const float* nvectx, const float* t0tx, const float* alatx, const float* doftx, 
        const float* ovecrx, const float* nvecrx, const float* alarx,
        const float c0, const int np, const float* pvec, float* pout
    )
    {
        int tpb, bpg, tid, itx, irx, ip;
        float tautx, apodtx, taurx, apodrx;

        // get cuda step sizes
        tpb = blockDim.x * blockDim.y * blockDim.z; // threads per block

        // Unique thread ID
        tid = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y;
        tid += tpb * blockIdx.x + tpb * blockIdx.y * gridDim.x + tpb * blockIdx.z * gridDim.x * gridDim.y;

        if (tid >= rfinfo.ntx * rfinfo.nrx * np) return;

        // calculate the transmit, recieve, and recon point indices for the thread we are working with
        itx = tid / (rfinfo.nrx * np);
        irx = (tid / np) % rfinfo.nrx;
        ip  = tid % np;

        calc_tautx_apodtx(
            rfinfo.ndim, 
            &ovectx[itx*rfinfo.ndim], 
            &nvectx[itx*rfinfo.ndim], 
            c0, t0tx[itx], 
            alatx[itx], 
            doftx[itx], 
            &pvec[ip*rfinfo.ndim],
            &tautx, &apodtx
        );

        calc_taurx_apodrx(
            rfinfo.ndim, 
            &ovecrx[irx*rfinfo.ndim], 
            &nvecrx[irx*rfinfo.ndim], 
            c0,
            alarx[irx], 
            &pvec[ip*rfinfo.ndim],
            &taurx, &apodrx
        );

        // If valid, add the beamformed and apodized value
        if (0 != apodtx * apodrx)
        {
            atomicAdd(
                &pout[ip], 
                apodtx * apodrx * cube_interp(rfinfo.tInfo.x0, rfinfo.tInfo.dx, rfinfo.tInfo.nx, &rfdata[itx*rfinfo.nrx*rfinfo.tInfo.nx + irx*rfinfo.tInfo.nx], tautx + taurx, 0.0)
            );
        } 
    }

    /**
     * das_bmode_rxseparate_cubic: beamform data keeping RX data separate
     * 
     * RF channel data parameters:
     *   rfinfo: information about the rfdata
     *   rfdata: grid of rf data in the shape ntx by nrx by nt (stored in tInfo)
     * 
     * Transmit parameters:
     *   ovectx: origin of each transmision
     *   nvectx: normal vector of each of the transmit events
     *   t0tx: the timepoint at which the wave is at ovectx in each transmit event
     *   alatx: acceptance angle in radians relative to each nvectx
     * 
     * Receive parameters:
     *   ovecrx: origin of the receive points
     *   nvecrx: normal vector of each receive element
     *   alarx:  angular acceptance for each element relative to nvecrx
     * 
     * Field parameters:
     *   c0: the homogeneos speed of sound in the medium
     *   np: the number of recon points
     *   pvec: the location of each recon point
     *   pout: a vector length p x rx for the output bmode
     */
    __global__ 
    void das_bmode_rxseparate_cubic(
        const struct RFInfo rfinfo, const float* rfdata, 
        const float* ovectx, const float* nvectx, const float* t0tx, const float* alatx, const float* doftx, 
        const float* ovecrx, const float* nvecrx, const float* alarx,
        const float c0, const int np, const float* pvec, float* pout
    )
    {
        int tpb, bpg, tid, itx, irx, ip;
        float tautx, apodtx, taurx, apodrx;

        // get cuda step sizes
        tpb = blockDim.x * blockDim.y * blockDim.z; // threads per block

        // Unique thread ID
        tid = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y;
        tid += tpb * blockIdx.x + tpb * blockIdx.y * gridDim.x + tpb * blockIdx.z * gridDim.x * gridDim.y;

        if (tid >= rfinfo.ntx * rfinfo.nrx * np) return;

        // calculate the transmit, recieve, and recon point indices for the thread we are working with
        itx = tid / (rfinfo.nrx * np);
        irx = (tid / np) % rfinfo.nrx;
        ip  = tid % np;

        calc_tautx_apodtx(
            rfinfo.ndim, 
            &ovectx[itx*rfinfo.ndim], 
            &nvectx[itx*rfinfo.ndim], 
            c0, t0tx[itx], 
            alatx[itx], 
            doftx[itx], 
            &pvec[ip*rfinfo.ndim],
            &tautx, &apodtx
        );

        calc_taurx_apodrx(
            rfinfo.ndim, 
            &ovecrx[irx*rfinfo.ndim], 
            &nvecrx[irx*rfinfo.ndim], 
            c0,
            alarx[irx], 
            &pvec[ip*rfinfo.ndim],
            &taurx, &apodrx
        );

        if (0 != apodtx * apodrx)
        {
            atomicAdd(
                &pout[irx * np + ip], 
                apodtx * apodrx * cube_interp(rfinfo.tInfo.x0, rfinfo.tInfo.dx, rfinfo.tInfo.nx, &rfdata[itx*rfinfo.nrx*rfinfo.tInfo.nx + irx*rfinfo.tInfo.nx], tautx + taurx, 0.0)
            );
        }
    }
}
