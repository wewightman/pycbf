#include "hip/hip_runtime.h"
extern "C" {
    __global__ void my_linterp(const float x0, const float dx, const int nx, const float* y, const float* xout, const int nxout, float* yout, float fill) 
    {
        int tid = blockDim.x * blockIdx.x + threadIdx.x;
        if (tid >= nxout) return;
                                
        float xn = x0 + dx * (nx-1);
        float xo = xout[tid];
        
        if (xo == xn) {
            yout[tid] = y[nx-1];
            return;
        } else if ((xo < x0) || (xo > xn)) {
            yout[tid] = fill;
            return;
        }
                                
        int ixo = (int) ((xo - x0)/dx);
        float xi = x0 + ixo * dx;
        float delta = (xo - xi)/dx;
        yout[tid] = (1-delta) * y[ixo] + delta * y[ixo+1];
    }

    float cube_interp(const float x0, const float dx, const int nx, const float* y, float xout, float fill) 
    {
        float xn = x0 + dx * (nx-1);
        
        if (xout == xn) return y[nx-1];
        else if ((xout < x0) || (xout > xn)) return fill;
                                
        int ixo = (int) ((xout- x0)/dx);

        float mm2, mm1, mp0, mp1, mp2, w0, w1, sp0, sp1, a, b, c, d, delta;

        // boundary conditions (bc)- first point
        if (ixo == 0) {
            mp0 = y[ixo+1] - y[ixo+0];
            mp1 = y[ixo+2] - y[ixo+1];

            sp0 = mp0;
            sp1 = (mp0 + mp1)/2;

        } 
        // bc - second point
        else if (ixo == 1) {
            mm1 = y[ixo+0] - y[ixo-1];
            mp0 = y[ixo+1] - y[ixo-0];
            mp1 = y[ixo+2] - y[ixo+1];
            mp2 = y[ixo+3] - y[ixo+2];

            sp0 = (mp0 + mm1)/2;

            if ((mm1 == mp0) && (mp0 == mp1) && (mp1 == mp2)) sp1 = 0;
            else {
                w0  = abs(mp2 - mp1) + abs(mp2 + mp1)/2;
                w1  = abs(mp0 - mm1) + abs(mp0 + mm1)/2;
                sp1 = (w0 * mp0 + w1 * mp1) / (w0 + w1);
            }
            

        } 
        // bc - third to last point
        else if (ixo == nx-3) {
            mm2 = y[ixo+-1] - y[ixo-2];
            mm1 = y[ixo+0] - y[ixo-1];
            mp0 = y[ixo+1] - y[ixo-0];
            mp1 = y[ixo+2] - y[ixo+1];

            if ((mm2 == mm1) && (mm1 == mp0) && (mp0 == mp1)) sp0 = 0;
            else {
                w0  = abs(mp1 - mp0) + abs(mp1 + mp0)/2;
                w1  = abs(mm1 - mm2) + abs(mm1 + mm2)/2;
                sp0 = (w0 * mm1 + w1*mp0) / (w0 + w1);
            }

            sp1 = (mp0 + mp1)/2;
        }
        // bc - second to last point
        else if (ixo == nx-2) {
            mm1 = y[ixo+0] - y[ixo-1];
            mp0 = y[ixo+1] - y[ixo+0];

            sp0 = (mm1+mp0)/2;

            sp1 = mp0;
        }
        // all other points
        else {
            mm2 = y[ixo-1] - y[ixo-2];
            mm1 = y[ixo+0] - y[ixo-1];
            mp0 = y[ixo+1] - y[ixo+0];
            mp1 = y[ixo+2] - y[ixo+1];
            mp2 = y[ixo+3] - y[ixo+2];

            if ((mm2 == mm1) && (mm1 == mp0) && (mp0 == mp1)) sp0 = 0;
            else {
                w0  = abs(mp1 - mp0) + abs(mp1 + mp0)/2;
                w1  = abs(mm1 - mm2) + abs(mm1 + mm2)/2;
                sp0 = (w0 * mm1 + w1*mp0) / (w0 + w1);
            }

            if ((mm1 == mp0) && (mp0 == mp1) && (mp1 == mp2)) sp1 = 0;
            else {
                w0  = abs(mp2 - mp1) + abs(mp2 + mp1)/2;
                w1  = abs(mp0 - mm1) + abs(mp0 + mm1)/2;
                sp1 = (w0 * mp0 + w1*mp1) / (w0 + w1);
            }
        }

        a = y[ixo];
        b = sp0;
        c = (3*mp0 - 2*sp0 - sp1)/dx;
        d = (sp0 + sp1 - 2*mp0)/(dx*dx);

        delta = xout - (x0 + dx * ixo);

        return a + b * delta + c * delta * delta + d * delta * delta * delta;
    }

    struct xInfo {
        float x0;
        float dx;
        int nx;
    };

    __global__ void my_cubeterp(const struct xInfo xinfo, const float* y, const float* xout, const int nxout, float* yout, float fill) 
    {
        int tid = blockDim.x * blockIdx.x + threadIdx.x;
        if (tid >= nxout) return;
                                
        yout[tid] = cube_interp(xinfo.x0, xinfo.dx, xinfo.nx, y, xout[tid], 0.0);
    }

    __global__ void copy_struct(const struct xInfo xinfo, float* yout)
    {
        yout[0] = xinfo.x0;
        yout[1] = xinfo.dx;
        yout[2] = (float) xinfo.nx;
    }

    struct RFInfo {
        int ntx;
        int nrx;
        int np;
        int ndim;
        struct xInfo tInfo;
    };

    __global__ void beamform(const struct RFInfo rfinfo, const float* rfdata, )
    {

    }
}
